#include "hip/hip_runtime.h"
#include <cuda_toolkit/hip/hip_vector_types.h>
#include <quadmap/device_image.cuh>
#include <quadmap/texture_memory.cuh>
#include <quadmap/match_parameter.cuh>
#include <quadmap/pixel_cost.cuh>
#include <ctime>

namespace quadmap
{
//function declear here!
void bp_extract(DeviceImage<PIXEL_COST> &image_cost_map, DeviceImage<float> &depth);
__global__ void cost_distribute(
    DeviceImage<PIXEL_COST> *l0_cost_devptr,
    DeviceImage<PIXEL_COST> *l1_cost_devptr);
__global__ void bp(
    DeviceImage<PIXEL_COST> *data_devptr,
    DeviceImage<PIXEL_COST> *lm_devptr,
    DeviceImage<PIXEL_COST> *rm_devptr,
    DeviceImage<PIXEL_COST> *up_devptr,
    DeviceImage<PIXEL_COST> *dm_devptr,
    bool A_set,
    int i_leverl);
__global__ void upsample(
    DeviceImage<PIXEL_COST> *l1_message_devptr,
    DeviceImage<PIXEL_COST> *l0_message_devptr);
__global__ void depth_extract(
    DeviceImage<PIXEL_COST> *data_devptr,
    DeviceImage<PIXEL_COST> *lm_devptr,
    DeviceImage<PIXEL_COST> *rm_devptr,
    DeviceImage<PIXEL_COST> *up_devptr,
    DeviceImage<PIXEL_COST> *dm_devptr,
    DeviceImage<float> *extracted_depth_devptr);

//function define here!
//we only optimize the cost at 16x16 and 32x32 level, for finer level, we only optimize at local patch and jump the corser level
//we start the optimize at image_level, and the cost map begins at image_level
void bp_extract(DeviceImage<PIXEL_COST> &image_cost_map, DeviceImage<float> &depth)
{
    const int width = image_cost_map.width;
    const int height = image_cost_map.height;
    const int hbp_level = 4;

    // 4 levels: 4x4, 8x8, 16x16, 32x32
    // corresponsible to level 0, 1, 2 ,3 ,4 , 5(this is only used for optimize)
    int hbp_iterate[4] = {4, 10, 10, 10}; // from fine to coarse 3 level

    int h_width[4]; //next four level
    int h_height[4]; //next four level

    h_width[0] = width;
    h_height[0] = height;

    for(int i = 1; i < hbp_level; i++)
    {
        h_width[i] = (h_width[i - 1] + 1) / 2;
        h_height[i] = (h_height[i - 1] + 1) / 2;
    }

    //create the hierarchical cost map
    DeviceImage<PIXEL_COST> *prycost_hostptr[4];
    prycost_hostptr[0] = &image_cost_map;
    for(int i = 1; i < hbp_level; i++)
    {
        prycost_hostptr[i] = new DeviceImage<PIXEL_COST>(h_width[i], h_height[i]);
    }

    dim3 hier_block;
    dim3 hier_grid;
    hier_block.z = 64;
    for(int i = 1; i < hbp_level; i++)
    {
        hier_grid.x = h_width[i];
        hier_grid.y = h_height[i];
        cost_distribute <<< hier_grid, hier_block>>>(
            prycost_hostptr[i - 1]->dev_ptr,
            prycost_hostptr[i]->dev_ptr);
        hipDeviceSynchronize();
    }

    //loopy bp on each level
    //create the message four dirs
    DeviceImage<PIXEL_COST> *message_hostptr[4];
    message_hostptr[0] = new DeviceImage<PIXEL_COST>(h_width[hbp_level - 1], h_height[hbp_level - 1]);
    message_hostptr[1] = new DeviceImage<PIXEL_COST>(h_width[hbp_level - 1], h_height[hbp_level - 1]);
    message_hostptr[2] = new DeviceImage<PIXEL_COST>(h_width[hbp_level - 1], h_height[hbp_level - 1]);
    message_hostptr[3] = new DeviceImage<PIXEL_COST>(h_width[hbp_level - 1], h_height[hbp_level - 1]);
    message_hostptr[0]->zero();
    message_hostptr[1]->zero();
    message_hostptr[2]->zero();
    message_hostptr[3]->zero();

    for(int i_leverl = hbp_level - 1; i_leverl >= 0; i_leverl--)
    {
        // /*if i_leverl is not the coarsest, initialize the message*/
        if( i_leverl < (hbp_level - 1) )
        {
            DeviceImage<PIXEL_COST> *message_next_hostptr[4];
            message_next_hostptr[0] = new DeviceImage<PIXEL_COST>(h_width[i_leverl], h_height[i_leverl]);
            message_next_hostptr[1] = new DeviceImage<PIXEL_COST>(h_width[i_leverl], h_height[i_leverl]);
            message_next_hostptr[2] = new DeviceImage<PIXEL_COST>(h_width[i_leverl], h_height[i_leverl]);
            message_next_hostptr[3] = new DeviceImage<PIXEL_COST>(h_width[i_leverl], h_height[i_leverl]);

            dim3 message_up_block;
            dim3 message_up_grid;
            message_up_block.x = 64;
            message_up_grid.x = h_width[i_leverl + 1];
            message_up_grid.y = h_height[i_leverl + 1];
            for(int mess_i = 0; mess_i < 4; mess_i++)
                upsample <<< message_up_grid, message_up_block>>>(
                    message_hostptr[mess_i]->dev_ptr,
                    message_next_hostptr[mess_i]->dev_ptr);

            hipDeviceSynchronize();

            for(int mess_i = 0; mess_i < 4; mess_i++)
            {
                delete message_hostptr[mess_i];
                message_hostptr[mess_i] = message_next_hostptr[mess_i];
            }
        }

        // /*loopy bp*/
        dim3 bp_block;
        dim3 bp_grid;
        bp_block.x = 4;
        bp_block.y = 64;
        bp_grid.x = h_width[i_leverl];
        bp_grid.y = h_height[i_leverl];
        bp_grid.x = (bp_grid.x + 1) / 2; //every iterate on the A or B set of the whole image
        bool A_set = true;
        for(int i_iterate = 0; i_iterate < hbp_iterate[i_leverl]; i_iterate++)
        {
            bp <<< bp_grid, bp_block>>>(
                prycost_hostptr[i_leverl]->dev_ptr,
                message_hostptr[0]->dev_ptr,
                message_hostptr[1]->dev_ptr,
                message_hostptr[2]->dev_ptr,
                message_hostptr[3]->dev_ptr,
                A_set,
                i_leverl + 2);
            A_set = !A_set;
            hipDeviceSynchronize();
        }
    }

    dim3 depth_extract_block;
    dim3 depth_extract_grid;
    depth_extract_block.x = DEPTH_NUM;
    depth_extract_grid.x = width;
    depth_extract_grid.y = height;
    depth_extract <<< depth_extract_grid, depth_extract_block>>>(
        prycost_hostptr[0]->dev_ptr,
        message_hostptr[0]->dev_ptr,
        message_hostptr[1]->dev_ptr,
        message_hostptr[2]->dev_ptr,
        message_hostptr[3]->dev_ptr,
        depth.dev_ptr);

    for(int i = 1; i < hbp_level; i++)
    {
        delete prycost_hostptr[i];
    }
    delete message_hostptr[0];
    delete message_hostptr[1];
    delete message_hostptr[2];
    delete message_hostptr[3];
}

__global__ void cost_distribute(DeviceImage<PIXEL_COST> *l0_cost_devptr,
                                DeviceImage<PIXEL_COST> *l1_cost_devptr)
{
    const int width = l1_cost_devptr->width;
    const int height = l1_cost_devptr->height;
    const int l0_width = l0_cost_devptr->width;
    const int l0_height = l0_cost_devptr->height;

    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int depth_id = threadIdx.z;

    if(x >= width || y >= height)
        return;

    float cost_sum(0.0f);

    for(int i = 0; i < 2; i++)
    {
        for(int j = 0; j < 2; j++)
        {
            if( (2 * x + i) < l0_width && (2 * y + j) < l0_height)
            {
                cost_sum += (l0_cost_devptr->atXY((2 * x + i), (2 * y + j))).get_cost(depth_id);
            }
        }
    }

    (l1_cost_devptr->atXY(x, y)).set_cost(depth_id, cost_sum);
}

__global__ void bp(
    DeviceImage<PIXEL_COST> *data_devptr,
    DeviceImage<PIXEL_COST> *lm_devptr,
    DeviceImage<PIXEL_COST> *rm_devptr,
    DeviceImage<PIXEL_COST> *up_devptr,
    DeviceImage<PIXEL_COST> *dm_devptr,
    bool A_set,
    int i_leverl)
{
    int x = blockIdx.x;
    int y = blockIdx.y;
    int dir = threadIdx.x;
    int depth_id = threadIdx.y;

    if(i_leverl <= 4)
    {
        int size = 1 << i_leverl;
        int pixel_level = tex2D(quadtree_tex, x * size, y * size);
        if(pixel_level > i_leverl)
            return;
    }

    float P1 = 0.003f;
    float P2 = 0.01f;

    if(A_set)
        x = x * 2 + y % 2;
    else
        x = x * 2 + (y + 1) % 2 ;

    const int width = data_devptr->width;
    const int height = data_devptr->height;
    if(x >= width || y >= height)
        return;

    bool on_left, on_right, on_up, on_down;
    on_left = on_right = on_up = on_down = false;

    if(x == 0)
        on_left = true;
    if(x == width - 1)
        on_right = true;
    if(y == 0)
        on_up = true;
    if(y == height - 1)
        on_down = true;

    __shared__ float neighbor_cost[4][DEPTH_NUM];
    __shared__ float neighbor_cost_min[4][DEPTH_NUM];
    __shared__ float raw_cost[4][DEPTH_NUM];

    neighbor_cost[dir][depth_id] = (data_devptr->atXY(x, y)).get_cost(depth_id);

    if(dir != 0 && !on_up) // to up
    {
        neighbor_cost[dir][depth_id] += (dm_devptr->atXY(x, y - 1)).get_cost(depth_id);
    }
    if(dir != 1 && !on_down) // to down
    {
        neighbor_cost[dir][depth_id] += (up_devptr->atXY(x, y + 1)).get_cost(depth_id);
    }
    if(dir != 2 && !on_left) // to left
    {
        neighbor_cost[dir][depth_id] += (rm_devptr->atXY(x - 1, y)).get_cost(depth_id);
    }
    if(dir != 3 && !on_right) // to right
    {
        neighbor_cost[dir][depth_id] += (lm_devptr->atXY(x + 1, y)).get_cost(depth_id);
    }
    neighbor_cost_min[dir][depth_id] = neighbor_cost[dir][depth_id];
    __syncthreads();

    //find min
    for(int i = DEPTH_NUM / 2; i > 0; i = i / 2)
    {
        if(depth_id < i && neighbor_cost_min[dir][depth_id + i] < neighbor_cost_min[dir][depth_id])
        {
            neighbor_cost_min[dir][depth_id] = neighbor_cost_min[dir][depth_id + i];
        }
        __syncthreads();
    }

    //find min cost for every message
    float min_cost = neighbor_cost[dir][depth_id];
    if(depth_id > 0)
        min_cost = fminf(min_cost, neighbor_cost[dir][depth_id - 1] + P1);
    if(depth_id < DEPTH_NUM - 1)
        min_cost = fminf(min_cost, neighbor_cost[dir][depth_id + 1] + P1);
    min_cost = fminf(min_cost, neighbor_cost_min[dir][0] + P2);

    raw_cost[dir][depth_id] = min_cost;
    __syncthreads();

    for(int i = DEPTH_NUM / 2; i > 0; i = i / 2)
    {
        if(depth_id < i)
        {
            raw_cost[dir][depth_id] += raw_cost[dir][depth_id + i];
        }
        __syncthreads();
    }

    min_cost = min_cost - raw_cost[dir][0] / (float) DEPTH_NUM;

    if(dir == 0) //up
        (up_devptr->atXY(x, y)).set_cost(depth_id, min_cost);
    else if(dir == 1) //to down
        (dm_devptr->atXY(x, y)).set_cost(depth_id, min_cost);
    else if(dir == 2) // to left
        (lm_devptr->atXY(x, y)).set_cost(depth_id, min_cost);
    else // to right
        (rm_devptr->atXY(x, y)).set_cost(depth_id, min_cost);
}

__global__ void depth_extract(
    DeviceImage<PIXEL_COST> *data_devptr,
    DeviceImage<PIXEL_COST> *lm_devptr,
    DeviceImage<PIXEL_COST> *rm_devptr,
    DeviceImage<PIXEL_COST> *up_devptr,
    DeviceImage<PIXEL_COST> *dm_devptr,
    DeviceImage<float> *extracted_depth_devptr)
{
    int x = blockIdx.x;
    int y = blockIdx.y;
    int width = data_devptr->width;
    int height = data_devptr->height;
    int depth_id = threadIdx.x;
    int pixel_level = tex2D(quadtree_tex, x * 4, y * 4);
    int level_size = 1 << pixel_level;
    if(x * 4 % level_size != 0 || y * 4 % level_size != 0)
        return;

    __shared__ float cost[DEPTH_NUM];
    __shared__ float min_cost[DEPTH_NUM];
    __shared__ int min_id[DEPTH_NUM];
    cost[depth_id] = data_devptr->atXY(x, y).get_cost(depth_id);
    if(x != 0)
        cost[depth_id] += rm_devptr->atXY(x - 1, y).get_cost(depth_id);
    if(x != width - 1)
        cost[depth_id] += lm_devptr->atXY(x + 1, y).get_cost(depth_id);
    if(y != 0)
        cost[depth_id] += dm_devptr->atXY(x, y - 1).get_cost(depth_id);
    if(y != height - 1)
        cost[depth_id] += up_devptr->atXY(x, y + 1).get_cost(depth_id);
    min_cost[depth_id] = cost[depth_id];
    min_id[depth_id] = depth_id;
    __syncthreads();
    for(int i = DEPTH_NUM / 2; i > 0; i /= 2)
    {
        if(depth_id < i && min_cost[depth_id + i] < min_cost[depth_id])
        {
            min_cost[depth_id] = min_cost[depth_id + i];
            min_id[depth_id] = min_id[depth_id + i];
        }
        __syncthreads();
    }

    if(depth_id == 0)
    {
        float disparity = min_id[0];
        if(min_id[0] > 0 && min_id[0] < DEPTH_NUM - 1)
        {
            float cost_pre = cost[min_id[0] - 1];
            float cost_post = cost[min_id[0] + 1];
            float a = cost_pre - 2.0f * min_cost[0] + cost_post;
            float b = - cost_pre + cost_post;
            float b_a = b/a;
            if(isfinite(b_a))
	            disparity = (float) min_id[0] - b_a / 2.0f;
        }
        extracted_depth_devptr->atXY(x * 4, y * 4) = 1.0 / (STEP_INV_DEPTH * disparity + MIN_INV_DEPTH);
    }
}

__global__ void upsample(
    DeviceImage<PIXEL_COST> *l1_message_devptr,
    DeviceImage<PIXEL_COST> *l0_message_devptr)
{
    const int depth_id = threadIdx.x;
    const int x = blockIdx.x; // in l1 image
    const int y = blockIdx.y; // in l1 image
    const int l0_width = l0_message_devptr->width;
    const int l0_height = l0_message_devptr->height;

    float value = (l1_message_devptr->atXY(x, y)).get_cost(depth_id);
    for(int j = 0; j < 2; j++)
    {
        for(int i = 0; i < 2; i++)
        {
            int x_up = x * 2 + i;
            int y_up = y * 2 + j;
            if(x_up < l0_width && y_up < l0_height)
                (l0_message_devptr->atXY(x_up, y_up)).set_cost(depth_id, value);
        }
    }
}
}