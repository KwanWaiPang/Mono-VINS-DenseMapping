#include "hip/hip_runtime.h"
#include <cuda_toolkit/hip/hip_vector_types.h>
#include <quadmap/device_image.cuh>
#include <quadmap/texture_memory.cuh>
#include <ctime>

namespace quadmap
{
//declear function
void generate_gradient(DeviceImage<float> &image, DeviceImage<float2> &gradient_map);
__global__ void gradient_kernel(DeviceImage<float> *image_dev_ptr, DeviceImage<float2> *gradient_dev_ptr);

//define function
void generate_gradient(DeviceImage<float> &image, DeviceImage<float2> &gradient_map)
{
	int width = gradient_map.width;
	int height = gradient_map.height;

	dim3 block;
	dim3 grid;
	block.x = 16;
	block.y = 16;
	grid.x = (width + block.x - 1) / block.x;
	grid.y = (height + block.y - 1) / block.y;
	gradient_kernel<<<grid, block>>>(image.dev_ptr, gradient_map.dev_ptr);
}
__global__ void gradient_kernel(DeviceImage<float> *image_dev_ptr, DeviceImage<float2> *gradient_dev_ptr)
{
	const int x = threadIdx.x + blockIdx.x * blockDim.x;
	const int y = threadIdx.y + blockIdx.y * blockDim.y;
	const int width = gradient_dev_ptr->width;
	const int height = gradient_dev_ptr->height;

	if (x >= width - 1 || y >= height - 1 || x <= 0 || y <= 0)
		return;

	float right_color = image_dev_ptr->atXY(x+1,y);
	float left_color = image_dev_ptr->atXY(x-1,y);
	float down_color = image_dev_ptr->atXY(x,y+1);
	float up_color = image_dev_ptr->atXY(x,y-1);

	gradient_dev_ptr->atXY(x, y) = make_float2((right_color - left_color)/2.0, (down_color - up_color)/2.0);
}
}